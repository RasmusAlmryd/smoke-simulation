#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

//#include <sys/time.h>
#include ""
#include "smokeSimulation.cuh"


void getGPUProperties() {
	//Discover GPU attributes
	hipError_t err;
	int devices;	//In case there are more than one GPUs
	hipDeviceProp_t prop;
	err = hipGetDeviceCount(&devices);
	if (!err) {
		for (int i = 0; i < devices; i++) {
			printf("CUDA Device - ID %d\n", i);
			err = hipGetDeviceProperties(&prop, i);
			if (!err) {
				printf("Max threads per block: 		%d\n", prop.maxThreadsPerBlock);
				printf("Max block dimensions: 		(%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
				printf("Max grid dimensions: 		(%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
				printf("Shared memory per block: 	%.2lfKB\n", prop.sharedMemPerBlock / 1024.);
			}
			printf("\n");
		}
	}
	else {
		fprintf(stderr, "Error finding available GPUs, now exiting\n");
		exit(-1);
	}
}


__global__ void smokeKernal(float *grid, int width, int height, int depth) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < width && y < height && z < depth) {
		if (y < height - 1) {
			if (grid[x + (y + 1) * width + z * width * height] > 0.0f) {
				grid[x + y * width + z * width * height] = 1.0f;
			}
			else {
				grid[x + y * width + z * width * height] = 0.0f;
			}
		}
		else {
			grid[x + y * width + z * width * height] = 0.0f;
		}
		
		//grid[4] = 1.0f;
		//grid[8] = 1.0f;
		//grid[27] = 1.0f;
	}
}

void updateGrid(float *grid, std::vector<int> grid_size, float delta) {
	hipError_t err;

	int width = grid_size[0]; //x
	int heigth = grid_size[1]; //y
	int depth = grid_size[2]; //z

	printf("width: %d, height: %d, depth: %d \n", width, heigth, depth);

	int size = width * heigth * depth * sizeof(float);

	float* dev_grid;

	err = hipMalloc(&dev_grid, size);
	if (err != 0) {
		fprintf(stderr, "Error allocating gird on GPU\n");
		exit(-1);
	}

	
 
	

	err = hipMemcpy(dev_grid, grid, size, hipMemcpyHostToDevice); 
	if (err != hipSuccess) { 
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err)); 
		exit(-1); 
	}

	dim3 dimBlock(8, 8, 8); //512 
	dim3 dimGrid(ceil(width / 8.0), ceil(heigth / 8.0), ceil(depth / 8.0)); 

	printf("width: %d, height: %d, depth: %d \n", dimGrid.x, dimGrid.y, dimGrid.z); 

	smokeKernal<<<dimGrid, dimBlock>>> (dev_grid, width, heigth, depth);

	// Copy data from device to host
	err = hipMemcpy(grid, dev_grid, size, hipMemcpyDeviceToHost);  
	if (err != hipSuccess) { 
		fprintf(stderr, "Error copying data from device to host: %s\n", hipGetErrorString(err)); 
		exit(-1); 
	}

	hipFree(dev_grid);

}