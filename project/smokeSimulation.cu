#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

//#include <sys/time.h>
#include ""
#include "smokeSimulation.cuh"


uint3 smokeDim;
uint3 smokeStaggeredDim;

int smokeIndex = 0; // (0,1)
float* dev_smoke[2];
float* dev_u[2];
float* dev_v[2];
float* dev_w[2];
bool* dev_s;
float* dev_p;

float gravity = -9.82; // m/s^2

#define OVER_RELAXATION 1.3;

void getGPUProperties() {
	//Discover GPU attributes
	hipError_t err;
	int devices;	//In case there are more than one GPUs
	hipDeviceProp_t prop;
	err = hipGetDeviceCount(&devices);
	if (!err) {
		for (int i = 0; i < devices; i++) {
			printf("CUDA Device - ID %d\n", i);
			err = hipGetDeviceProperties(&prop, i);
			if (!err) {
				printf("Max threads per block: 		%d\n", prop.maxThreadsPerBlock);
				printf("Max block dimensions: 		(%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
				printf("Max grid dimensions: 		(%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
				printf("Shared memory per block: 	%.2lfKB\n", prop.sharedMemPerBlock / 1024.);
			}
			printf("\n");
		}
	}
	else {
		fprintf(stderr, "Error finding available GPUs, now exiting\n");
		exit(-1);
	}
}

void initializeVolume(float* smoke_grid, unsigned int width, unsigned int heigth, unsigned int depth) {
	hipError_t err; 

	smokeDim = { width, heigth, depth };
	int smokeSize = width * heigth * depth * sizeof(float);

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_smoke[i], smokeSize);
		if (err != 0) {
			fprintf(stderr, "Error allocating smoke gird on GPU\n");
			exit(-1);
		}
	}

	err = hipMemcpy(dev_smoke[0], smoke_grid, smokeSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}


	/*
	*	should be different for each velocity axis.
	*	ex u size: x: dim.x+1, y: dim.y, z: dim.z
	*/
	int velSize = (width+1) * (heigth+1) * (depth+1) * sizeof(float); 
	smokeStaggeredDim = { (width + 1) , (heigth + 1) , (depth + 1) };

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_u[i], velSize);
		if (err != 0) { 
			fprintf(stderr, "Error allocating U gird on GPU\n"); 
			exit(-1); 
		}
	}

	err = hipMemset(dev_u[0], 0, velSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_v[i], velSize);
		if (err != 0) {
			fprintf(stderr, "Error allocating V gird on GPU\n");
			exit(-1);
		}
	}

	err = hipMemset(dev_v[0], 0, velSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_w[i], velSize);
		if (err != 0) {
			fprintf(stderr, "Error allocating V gird on GPU\n");
			exit(-1);
		}
	}

	err = hipMemset(dev_w[0], 0, velSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}


	// generate smoke bounds
	int sSize = width * heigth * depth * sizeof(bool);
	err = hipMalloc(&dev_s, sSize);
	if (err != 0) {
		fprintf(stderr, "Error allocating S gird on GPU\n");
		exit(-1);
	}

	std::vector<unsigned char> s_host(width * heigth * depth, 1);
	for (int x = 0; x < width; x++) {
		for (int z = 0; z < depth; z++) {
			int y = 0;
			s_host[x + y * width + z * width * heigth] = 0;
		}
	}

	err = hipMemcpy(dev_s, s_host.data(), sSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	s_host.clear();


	// Presuare 
	int pSize = width * heigth * depth * sizeof(float);
	err = hipMalloc(&dev_p, pSize);
	if (err != 0) {
		fprintf(stderr, "Error allocating S gird on GPU\n");
		exit(-1);
	}

	/*err = hipMemset(dev_p, st.data(), sSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}*/
	/*err = hipMemset(dev_s, 1, sSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err)); 
		exit(-1); 
	}*/
}

void deleteVolume() {
	for (int i = 0; i < 2; i++) {
		hipFree(dev_smoke[i]);
		hipFree(dev_u[i]);
		hipFree(dev_v[i]);
	}
	hipFree(dev_s);
}

__global__ void integrate(float *v, bool *s, uint3 normalDim, uint3 staggeredDim, float dt, float gravity) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y+1;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[x + (y - 1) * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += gravity * dt;
		}
	}
}

__global__ void divergence(float* u, float* v, float* w, float *p, bool* s, uint3 normalDim, uint3 staggeredDim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x+1;
	int y = threadIdx.y + blockDim.y * blockIdx.y+1;
	int z = threadIdx.z + blockDim.z * blockIdx.z+1;

	if (x < normalDim.x-1 && y < normalDim.y-1 && z < normalDim.z-1) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] == 0)
			return;

		int acc_s = 0;
		int sx1 = s[(x+1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sx0 = s[(x-1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sy1 = s[x + (y+1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sy0 = s[x + (y-1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sz1 = s[x + y * normalDim.x + (z+1) * normalDim.x * normalDim.y];
		int sz0 = s[x + y * normalDim.x + (z-1) * normalDim.x * normalDim.y];
		acc_s = sx0 + sx1 + sy0 + sy1 + sz0 + sz1;

		if (acc_s == 0) return;

		float div = -u[(x) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
			-v[x + (y) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
			-w[x + y * staggeredDim.x + (z) * staggeredDim.x * staggeredDim.y] + w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y];

		/*div /= acc_s;
		div *= OVER_RELAXATION;*/

		float p_t = (-div / acc_s) * OVER_RELAXATION;
		p[x + y * normalDim.x + z * normalDim.x * normalDim.y] = p_t;

		/*u[(x    ) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sx0);
		u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sx1);
		v[x + (y    ) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sy0);
		v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sy1);
		w[x + y * staggeredDim.x + (z    ) * staggeredDim.x * staggeredDim.y] -= (p_t * sz0);
		w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y] += (p_t * sz1);*/


	}
}

__global__ void diverge(float* u, float* v, float* w, float* p, bool* s, uint3 normalDim, uint3 staggeredDim) {

	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x - 1 && y < normalDim.y - 1 && z < normalDim.z - 1) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] == 0)
			return;

		int sx1 = s[(x + 1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sx0 = s[(x - 1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sy1 = s[x + (y + 1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sy0 = s[x + (y - 1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sz1 = s[x + y * normalDim.x + (z + 1) * normalDim.x * normalDim.y];
		int sz0 = s[x + y * normalDim.x + (z - 1) * normalDim.x * normalDim.y];

		float p_t = p[x + y * normalDim.x + z * normalDim.x * normalDim.y];
		
		u[(x + 0) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sx0);
		u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sx1);
		v[x + (y + 0) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sy0);
		v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sy1);
		w[x + y * staggeredDim.x + (z + 0) * staggeredDim.x * staggeredDim.y] -= (p_t * sz0);
		w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y] += (p_t * sz1);
	}
}

//staggered for 
__global__ void divergenceTemp(float* u, float* v, float* w, bool* s, uint3 normalDim, uint3 staggeredDim, char offset) {

	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	x = x * 2 - (y + z + offset) % 2;

	if (x < normalDim.x - 1 && y < normalDim.y - 1 && z < normalDim.z - 1) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] == 0)
			return;

		int acc_s = 0;
		int sx1 = s[(x + 1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sx0 = s[(x - 1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sy1 = s[x + (y + 1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sy0 = s[x + (y - 1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sz1 = s[x + y * normalDim.x + (z + 1) * normalDim.x * normalDim.y];
		int sz0 = s[x + y * normalDim.x + (z - 1) * normalDim.x * normalDim.y];
		acc_s = sx0 + sx1 + sy0 + sy1 + sz0 + sz1;

		if (acc_s == 0) return;

		float div = -u[(x)+y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
			-v[x + (y)*staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
			-w[x + y * staggeredDim.x + (z)*staggeredDim.x * staggeredDim.y] + w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y];

		/*div /= acc_s;
		div *= OVER_RELAXATION;*/

		float p_t = (-div / acc_s) * OVER_RELAXATION;
		//p[x + y * normalDim.x + z * normalDim.x * normalDim.y] = p_t;

		u[(x + 0) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sx0);
		u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sx1);
		v[x + (y + 0) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sy0);
		v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sy1);
		w[x + y * staggeredDim.x + (z + 0) * staggeredDim.x * staggeredDim.y] -= (p_t * sz0);
		w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y] += (p_t * sz1);

	}
}

__global__ void extrapolate(float* u, float* v, float* w, uint3 normalDim, uint3 staggeredDim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;


	if (x == 0){
		u[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
	}

}


__device__ float avgU(float* vel, int x, int y, int z, uint3 staggeredDim) {
	float u = 
		vel[x + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
	return u / 8;
}

__device__ float avgV(float* vel, int x, int y, int z, uint3 staggeredDim) {
	float v = 
		vel[x + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + (y + 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y + 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];

	return v / 8;
}

__device__ float avgW(float* vel, int x, int y, int z, uint3 staggeredDim) {
	float w =
		vel[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y];
	return w / 8;
}



__device__ float sampleSmoke(float* field, uint3 gridDim, float3 pos, float3 delta, uint3 normalDim) {
	
	float x = fmaxf(fminf(pos.x, normalDim.x-1), 1);
	float y = fmaxf(fminf(pos.y, normalDim.y-1), 1);
	float z = fmaxf(fminf(pos.z, normalDim.z-1), 1);

	//interpelation
	int x0 = (int)fminf(floorf(x - delta.x), normalDim.x-1);
	float xw1 = (x - delta.x) - x0;
	float xw0 = 1 - xw1;
	int x1 = (int)fminf(x0 + 1, normalDim.x - 1);

	int y0 = (int)fminf(floorf(y - delta.y), normalDim.y-1);
	float yw1 = (y - delta.y) - y0;
	float yw0 = 1 - yw1;
	int y1 = (int)fminf(y0 + 1, normalDim.y - 1);

	int z0 = (int)fminf(floorf(z - delta.z), normalDim.z-1);
	float zw1 = (z - delta.z) - z0;
	float zw0 = 1 - zw1;
	int z1 = (int)fminf(z0 + 1, normalDim.z - 1);

	return
		xw0 * yw0 * zw0 * field[x0 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw0 * field[x1 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw0 * field[x0 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw0 * field[x1 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw0 * zw1 * field[x0 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw1 * field[x1 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw1 * field[x0 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw1 * field[x1 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y];
		

}

__device__ float sampleVelocity(float* field, uint3 gridDim, float3 pos, float3 delta, uint3 normalDim) {

	float x = fmaxf(fminf(pos.x, gridDim.x - 1), 1);
	float y = fmaxf(fminf(pos.y, gridDim.y - 1), 1);
	float z = fmaxf(fminf(pos.z, gridDim.z - 1), 1);

	//interpelation
	int x0 = (int)fminf(floorf(x - delta.x), gridDim.x - 1);
	float xw1 = (x - delta.x) - x0;
	float xw0 = 1 - xw1;
	int x1 = (int)fminf(x0 + 1, gridDim.x - 1);

	int y0 = (int)fminf(floorf(y - delta.y), gridDim.y - 1);
	float yw1 = (y - delta.y) - y0;
	float yw0 = 1 - yw1;
	int y1 = (int)fminf(y0 + 1, gridDim.y - 1);

	int z0 = (int)fminf(floorf(z - delta.z), gridDim.z - 1);
	float zw1 = (z - delta.z) - z0;
	float zw0 = 1 - zw1;
	int z1 = (int)fminf(z0 + 1, gridDim.z - 1);

	return
		xw0 * yw0 * zw0 * field[x0 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw0 * field[x1 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw0 * field[x0 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw0 * field[x1 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw0 * zw1 * field[x0 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw1 * field[x1 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw1 * field[x0 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw1 * field[x1 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y];


}





__global__ void velocityAdvectionU(float *u0, float *u1, float *v0, float *w0, bool* s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x && y < normalDim.y-1 && z < normalDim.z-1) {
		
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[(x - 1) + y * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			float x_t = x;
			float y_t = y + 0.5;
			float z_t = z + 0.5; 

			float u = u0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
			float v = avgV(v0, x, y, z, staggeredDim);
			float w = avgW(w0, x, y, z, staggeredDim);

			x_t = x_t - dt * u;
			y_t = y_t - dt * v;
			z_t = z_t - dt * w;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0, 0.5, 0.5 };

			float newU = sampleSmoke(u0, staggeredDim, pos, delta, normalDim);
			u1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = newU;

		}
	}
}

__global__ void velocityAdvectionV(float* v0, float* v1, float* u0, float* w0, bool* s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x-1 && y < normalDim.y && z < normalDim.z - 1) {

		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[x + (y-1) * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			float x_t = x + 0.5;
			float y_t = y;
			float z_t = z + 0.5;

			float u = avgU(u0, x, y, z, staggeredDim);
			float v = v0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
			float w = avgW(w0, x, y, z, staggeredDim);

			x_t = x_t - dt * u;
			y_t = y_t - dt * v;
			z_t = z_t - dt * w;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0.5, 0, 0.5 };

			float newU = sampleSmoke(v0, staggeredDim, pos, delta, normalDim);
			v1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = newU;

		}
	}
}

__global__ void velocityAdvectionW(float* w0, float* w1, float* u0, float* v0, bool* s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x - 1 && y < normalDim.y - 1 && z < normalDim.z) {

		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[x + y * normalDim.x + (z-1) * normalDim.x * normalDim.y] != 0) {
			float x_t = x + 0.5;
			float y_t = y + 0.5;
			float z_t = z;

			float u = avgU(u0, x, y, z, staggeredDim);
			float v = avgV(v0, x, y, z, staggeredDim);
			float w = w0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];

			x_t = x_t - dt * u;
			y_t = y_t - dt * v;
			z_t = z_t - dt * w;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0.5, 0.5, 0 };

			float newU = sampleSmoke(w0, staggeredDim, pos, delta, normalDim);
			w1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = newU;

		}
	}
}

__global__ void advectSmoke(float* smoke0, float *smoke1, float* u, float* v, float* w, bool *s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x+1;
	int y = threadIdx.y + blockDim.y * blockIdx.y+1;
	int z = threadIdx.z + blockDim.z * blockIdx.z+1;

	if (x < normalDim.x-1 && y < normalDim.y-1 && z < normalDim.z-1) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			float u_t = (u[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y]) / 2;
			float v_t = (v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y]) / 2;
			float w_t = (w[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y]) / 2;

			float x_t = x + 0.5 - u_t * dt;
			float y_t = y + 0.5 - v_t * dt;
			float z_t = z + 0.5 - w_t * dt;
			/*float x_t = x + 0.5 ;
			float y_t = y + 0.5 + 1;
			float z_t = z + 0.5 ;*/

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0.5, 0.5, 0.5 };

			smoke1[x + y * normalDim.x + z * normalDim.x * normalDim.y] = sampleSmoke(smoke0, normalDim, pos, delta, normalDim); //smoke0[x + (y+1) * normalDim.x + z * normalDim.x * normalDim.y];
		}
	}
}

__global__ void visualizeV(float* v, float* smoke, uint3 normalDim, uint3 staggeredDim, uint3 dir) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] =
			(v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[(x+dir.x) + (y + dir.y) * staggeredDim.x + (z+dir.z) * staggeredDim.x * staggeredDim.y]) / 2;
	}
}

__global__ void visualizeS(bool *s, float* smoke, uint3 normalDim, uint3 staggeredDim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] = (float)s[x + y * normalDim.x + z * normalDim.x * normalDim.y];
	}
}


__global__ void visualizeP(float *p, float* smoke, uint3 normalDim, uint3 staggeredDim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] = p[x + y * normalDim.x + z * normalDim.x * normalDim.y];
	}
}


__global__ void smokeKernal(float *grid, int width, int height, int depth) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < width && y < height && z < depth) {
		if (y < height - 1) {
			if (grid[x + (y + 1) * width + z * width * height] > 0.0f) {
				grid[x + y * width + z * width * height] = 1.0f;
			}
			else {
				grid[x + y * width + z * width * height] = 0.0f;
			}
		}
		else {
			grid[x + y * width + z * width * height] = 0.0f;
		}
		
		//grid[4] = 1.0f;
		//grid[8] = 1.0f;
		//grid[27] = 1.0f;
	}
}

int tempIndexNow = 1;
int tempIndexPast = 0;

void simulate(float* smoke_grid, float dt) {
	printf("smoke sim simulate()");
	hipError_t err;

	//return;
	tempIndexPast = tempIndexNow;
	if (tempIndexNow == 0) tempIndexNow = 1;
	else tempIndexNow = 0;

	dim3 dimBlock(8, 8, 8); //512 
	dim3 dimGrid(ceil(smokeDim.x / 8.0), ceil(smokeDim.y / 8.0), ceil(smokeDim.z / 8.0));

	//printf("width: %d, height: %d, depth: %d \n", dimGrid.x, dimGrid.y, dimGrid.z);
	printf("now: %d, past: %d", tempIndexNow, tempIndexPast);

	integrate<<<dimGrid, dimBlock>>>(dev_v[smokeIndex], dev_s, smokeDim, smokeStaggeredDim, dt, gravity);
	
	/*int num_iterations = 20;
	for (int i = 0; i < num_iterations; i++) {
		divergence << <dimGrid, dimBlock >> > (dev_u[smokeIndex], dev_v[smokeIndex], dev_w[smokeIndex], dev_p, dev_s, smokeDim, smokeStaggeredDim);
		diverge << <dimGrid, dimBlock >> > (dev_u[smokeIndex], dev_v[smokeIndex], dev_w[smokeIndex], dev_p, dev_s, smokeDim, smokeStaggeredDim);
	}*/

	dim3 divdimBlock(8, 8, 8); //512 
	dim3 divdimGrid(ceil(ceil(smokeDim.x / 2.0) / 8.0), ceil(smokeDim.y / 8.0), ceil(smokeDim.z / 8.0));
	int num_iterations = 50;
	for (int i = 0; i < num_iterations; i++) {
		divergenceTemp << <divdimGrid, divdimBlock >> > (dev_u[tempIndexNow], dev_v[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, 0);
		divergenceTemp << <divdimGrid, divdimBlock >> > (dev_u[tempIndexNow], dev_v[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, 1);
	}


	velocityAdvectionU << <dimGrid, dimBlock >> > (dev_u[tempIndexNow], dev_u[tempIndexPast], dev_v[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt);
	velocityAdvectionV << <dimGrid, dimBlock >> > (dev_v[tempIndexNow], dev_v[tempIndexPast], dev_u[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt);
	velocityAdvectionW << <dimGrid, dimBlock >> > (dev_w[tempIndexNow], dev_w[tempIndexPast], dev_u[tempIndexNow], dev_v[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt);

	advectSmoke << <dimGrid, dimBlock >> > (dev_smoke[tempIndexNow], dev_smoke[tempIndexPast], dev_u[tempIndexPast], dev_v[tempIndexPast], dev_w[tempIndexPast], dev_s, smokeDim, smokeStaggeredDim, dt);

	uint3 dir = { 0,0,1 };
	//visualizeV << <dimGrid, dimBlock >> > (dev_w[tempIndexPast], dev_smoke[smokeIndex], smokeDim, smokeStaggeredDim, dir);
	//visualizeS << <dimGrid, dimBlock >> > (dev_s, dev_smoke[smokeIndex], smokeDim, smokeStaggeredDim);
	//visualizeP << <dimGrid, dimBlock >> > (dev_p, dev_smoke[smokeIndex], smokeDim, smokeStaggeredDim);

	//err = hipMemcpy(smoke_grid, dev_smoke[smokeIndex], smokeDim.x * smokeDim.y * smokeDim.z * sizeof(float), hipMemcpyDeviceToHost);
	err = hipMemcpy(smoke_grid, dev_smoke[tempIndexPast], smokeDim.x * smokeDim.y * smokeDim.z * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from device to host: %s\n", hipGetErrorString(err));
		exit(-1);
	}
}

void updateGrid(float *grid, std::vector<int> grid_size, float delta) {
	hipError_t err;

	int width = grid_size[0]; //x
	int heigth = grid_size[1]; //y
	int depth = grid_size[2]; //z

	printf("width: %d, height: %d, depth: %d \n", width, heigth, depth);

	int size = width * heigth * depth * sizeof(float);

	float* dev_grid;

	err = hipMalloc(&dev_grid, size);
	if (err != 0) {
		fprintf(stderr, "Error allocating gird on GPU\n");
		exit(-1);
	}

	
 
	

	err = hipMemcpy(dev_grid, grid, size, hipMemcpyHostToDevice); 
	if (err != hipSuccess) { 
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err)); 
		exit(-1); 
	}

	dim3 dimBlock(8, 8, 8); //512 
	dim3 dimGrid(ceil(width / 8.0), ceil(heigth / 8.0), ceil(depth / 8.0)); 

	printf("width: %d, height: %d, depth: %d \n", dimGrid.x, dimGrid.y, dimGrid.z); 

	smokeKernal<<<dimGrid, dimBlock>>> (dev_grid, width, heigth, depth);

	// Copy data from device to host
	err = hipMemcpy(grid, dev_grid, size, hipMemcpyDeviceToHost);  
	if (err != hipSuccess) { 
		fprintf(stderr, "Error copying data from device to host: %s\n", hipGetErrorString(err)); 
		exit(-1); 
	}

	hipFree(dev_grid);

}