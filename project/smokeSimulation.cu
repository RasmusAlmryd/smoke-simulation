#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

//#include <sys/time.h>
#include ""
#include "smokeSimulation.cuh"

#include <glm/glm.hpp>
#include <glm/gtx/transform.hpp>
using namespace glm;


uint3 smokeDim;
uint3 smokeStaggeredDim;

int smokeIndex = 0; // (0,1)
float* dev_smoke[2];
float* dev_u[2];
float* dev_v[2];
float* dev_w[2];
bool* dev_s;



float gravity = -9.82; // m/s^2
float buoyancy_alpha = 2.0f;

float* getBuoyancy() {
	return &buoyancy_alpha;
}
float* getGravity() {
	return &gravity;
}

#define OVER_RELAXATION 1.9
#define MAX_VELOCITY_PER_STEP 3.0f //squared

float* dev_obstacles;
float* dev_smokeSources; 


struct Sphere {
	int id;
	float3 pos;
	float3 vel;
	float radius;
	short type; // 0: obstacle, 1:smoke 

};

std::vector<Sphere> objects;

int max_objects = 3;

int currentId = 0;



void getGPUProperties() {
	//Discover GPU attributes
	hipError_t err;
	int devices;	//In case there are more than one GPUs
	hipDeviceProp_t prop;
	err = hipGetDeviceCount(&devices);
	if (!err) {
		for (int i = 0; i < devices; i++) {
			printf("CUDA Device - ID %d\n", i);
			err = hipGetDeviceProperties(&prop, i);
			if (!err) {
				printf("Max threads per block: 		%d\n", prop.maxThreadsPerBlock);
				printf("Max block dimensions: 		(%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
				printf("Max grid dimensions: 		(%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
				printf("Shared memory per block: 	%.2lfKB\n", prop.sharedMemPerBlock / 1024.);
			}
			printf("\n");
		}
	}
	else {
		fprintf(stderr, "Error finding available GPUs, now exiting\n");
		exit(-1);
	}
}


int addObstacle(float x, float y, float z, float vx, float vy, float vz, float r) {
	float3 pos = { x,y,z };
	float3 vel = { vx,vy,vz };
	Sphere s = { currentId, pos, vel, r, 0};
	objects.push_back(s);
	currentId++;
	return currentId - 1;
}

int addSmokeSource(float x, float y, float z, float r) {
	float3 pos = { x,y,z };
	float3 vel = { 0,0,0 };
	Sphere s = { currentId, pos, vel, r, 1 };
	objects.push_back(s);
	currentId++;
	return currentId - 1;
}

void updateObjectPos(int id, float x, float y, float z) {
	float3 pos = { x,y,z };
	objects[id].pos = pos;
}

std::vector<Sphere> getObjectsOfType(int type) {
	std::vector<Sphere> object;
	for (int i = 0; i < objects.size(); i++) {
		if (objects[i].type == type) {
			object.push_back(objects[i]);
		}
	}

	return object;
}



void initializeVolume(float* smoke_grid, unsigned int width, unsigned int heigth, unsigned int depth) {
	hipError_t err; 

	smokeDim = { width, heigth, depth };
	int smokeSize = width * heigth * depth * sizeof(float);

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_smoke[i], smokeSize);
		if (err != 0) {
			fprintf(stderr, "Error allocating smoke gird on GPU\n");
			exit(-1);
		}
	}

	err = hipMemcpy(dev_smoke[0], smoke_grid, smokeSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}


	/*
	*	should be different for each velocity axis.
	*	ex u size: x: dim.x+1, y: dim.y, z: dim.z
	*/
	int velSize = (width+1) * (heigth+1) * (depth+1) * sizeof(float); 
	smokeStaggeredDim = { (width + 1) , (heigth + 1) , (depth + 1) };

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_u[i], velSize);
		if (err != 0) { 
			fprintf(stderr, "Error allocating U gird on GPU\n"); 
			exit(-1); 
		}
	}

	err = hipMemset(dev_u[0], 0, velSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_v[i], velSize);
		if (err != 0) {
			fprintf(stderr, "Error allocating V gird on GPU\n");
			exit(-1);
		}
	}

	err = hipMemset(dev_v[0], 0, velSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	for (int i = 0; i < 2; i++) {
		err = hipMalloc(&dev_w[i], velSize);
		if (err != 0) {
			fprintf(stderr, "Error allocating V gird on GPU\n");
			exit(-1);
		}
	}

	err = hipMemset(dev_w[0], 0, velSize);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}


	// generate smoke bounds
	int sSize = width * heigth * depth * sizeof(bool);
	err = hipMalloc(&dev_s, sSize);
	if (err != 0) {
		fprintf(stderr, "Error allocating S gird on GPU\n");
		exit(-1);
	}

	std::vector<unsigned char> s_host(width * heigth * depth, 1);
	int y = 0;
	for (int x = 0; x < width; x++) {
		for (int z = 0; z < depth; z++) {
			
			s_host[x + y * width + z * width * heigth] = 0;
		}
	}

	err = hipMemcpy(dev_s, s_host.data(), sSize, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	s_host.clear();




	// generate arrays for objects
	int obstacleSize = (3 + 3 + 1) * sizeof(float) * max_objects; //pos, vel, radius
	err = hipMalloc(&dev_obstacles, obstacleSize);
	if (err != 0) {
		fprintf(stderr, "Error allocating dev_obstacles gird on GPU\n");
		exit(-1);
	}

	int smokeSourcesSize = (3 + 1) * sizeof(float) * max_objects; //pos, radius
	err = hipMalloc(&dev_smokeSources, smokeSourcesSize);
	if (err != 0) {
		fprintf(stderr, "Error allocating dev_smokeSources gird on GPU\n");
		exit(-1);
	}



	float3 spherePosition = { smokeDim.x / 2.0, smokeDim.y / 2.0 , smokeDim.z / 2.0 };
}

void deleteVolume() {
	for (int i = 0; i < 2; i++) {
		hipFree(dev_smoke[i]);
		hipFree(dev_u[i]);
		hipFree(dev_v[i]);
		hipFree(dev_w[i]);
	}
	hipFree(dev_s);
}


__global__ void fillSmoke(float* smoke0, float* smoke1, uint3 normalDim, float* dev_smokeSources, int num_sources) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x-1 && y < normalDim.y-1 && z < normalDim.z-1) {
		
		for (int i = 0; i < num_sources; i++) {
			float sx = dev_smokeSources[i * 4 + 0];
			float sy = dev_smokeSources[i * 4 + 1];
			float sz = dev_smokeSources[i * 4 + 2];
			float sr = dev_smokeSources[i * 4 + 3];
			
			
			float dist = powf(x - sx, 2) + powf(y - sy, 2) + powf(z - sz, 2);
			if (dist < sr*sr) {
				smoke0[x + y * normalDim.x + z * normalDim.x * normalDim.y] = 1.0f;
				smoke1[x + y * normalDim.x + z * normalDim.x * normalDim.y] = 1.0f;
			}
		}

	}
}

__global__ void fillSphere(float* v0, float* v1, float value,  uint3 staggeredDim, float3 sphereCenter, float sphereDiameter) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < staggeredDim.x - 1 && y < staggeredDim.y - 1 && z < staggeredDim.z - 1) {
		float dist = powf(x - sphereCenter.x, 2) + powf(y - sphereCenter.y, 2) + powf(z - sphereCenter.z, 2);
		if (dist < sphereDiameter) {
			v0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = value;
			v1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = value;
		}
	}
}

__global__ void fillObstacle(bool* s, uint3 normalDim, float* dev_obstacles, int num_obstacles) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x - 1 && y < normalDim.y - 1 && z < normalDim.z - 1) {
		for (int i = 0; i < num_obstacles; i++) {
			float sx = dev_obstacles[i * 7 + 0];
			float sy = dev_obstacles[i * 7 + 1];
			float sz = dev_obstacles[i * 7 + 2];
			float svx = dev_obstacles[i * 7 + 3];
			float svy = dev_obstacles[i * 7 + 4];
			float svz = dev_obstacles[i * 7 + 5];
			float sr = dev_obstacles[i * 7 + 6];
			
			float dist = powf(x - sx, 2) + powf(y - sy, 2) + powf(z - sz, 2);
			if (dist < sr*sr) {
				s[x + y * normalDim.x + z * normalDim.x * normalDim.y] = 0;
			}
			else {
				s[x + y * normalDim.x + z * normalDim.x * normalDim.y] = 1;
			}
		}
	}
}

__global__ void integrate(float *v, float *smoke, bool *s, uint3 normalDim, uint3 staggeredDim, float dt, float gravity, float buoyancy_alpha) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y+1;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[x + (y - 1) * normalDim.x + z * normalDim.x * normalDim.y] != 0) {

			float buoyancy = buoyancy_alpha* (smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] - 0.0f);
			v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] * gravity * dt + buoyancy * dt;
			//v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +=  gravity * dt + buoyancy * dt;

		}
	}
}

__global__ void velocityConfinement(float* u, float* v, float* w, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x +1;
	int y = threadIdx.y + blockDim.y * blockIdx.y +1;
	int z = threadIdx.z + blockDim.z * blockIdx.z +1;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		int staggeredIndex = x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y;

		float u_t = u[staggeredIndex];
		float v_t = v[staggeredIndex];
		float w_t = w[staggeredIndex];

		float velLength = u_t * u_t + v_t * v_t + w_t * w_t;

		if (velLength * dt > MAX_VELOCITY_PER_STEP* MAX_VELOCITY_PER_STEP) {
			u[staggeredIndex] = u_t * (MAX_VELOCITY_PER_STEP * MAX_VELOCITY_PER_STEP / (velLength * dt));
			v[staggeredIndex] = v_t * (MAX_VELOCITY_PER_STEP * MAX_VELOCITY_PER_STEP / (velLength * dt));
			w[staggeredIndex] = w_t * (MAX_VELOCITY_PER_STEP * MAX_VELOCITY_PER_STEP /(velLength * dt));
		}

	}
}



__global__ void divergence(float* u, float* v, float* w, bool* s, uint3 normalDim, uint3 staggeredDim, char offset) {

	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	x = x * 2 - (y + z + offset) % 2;

	if (x < normalDim.x - 1 && y < normalDim.y - 1 && z < normalDim.z - 1) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] == 0)
			return;

		int acc_s = 0;
		int sx1 = s[(x + 1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sx0 = s[(x - 1) + y * normalDim.x + z * normalDim.x * normalDim.y];
		int sy1 = s[x + (y + 1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sy0 = s[x + (y - 1) * normalDim.x + z * normalDim.x * normalDim.y];
		int sz1 = s[x + y * normalDim.x + (z + 1) * normalDim.x * normalDim.y];
		int sz0 = s[x + y * normalDim.x + (z - 1) * normalDim.x * normalDim.y];
		acc_s = sx0 + sx1 + sy0 + sy1 + sz0 + sz1;

		if (acc_s == 0) return;

		float div = -u[(x)+y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
			-v[x + (y)*staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
			-w[x + y * staggeredDim.x + (z)*staggeredDim.x * staggeredDim.y] + w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y];


		float p_t = (-div / acc_s) * OVER_RELAXATION;

		u[(x + 0) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sx0);
		u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sx1);
		v[x + (y + 0) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] -= (p_t * sy0);
		v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] += (p_t * sy1);
		w[x + y * staggeredDim.x + (z + 0) * staggeredDim.x * staggeredDim.y] -= (p_t * sz0);
		w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y] += (p_t * sz1);

	}
}

__global__ void extrapolate(float* velFeild, uint3 normalDim, uint3 staggeredDim, uchar3 direction) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	
	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		velFeild[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = velFeild[(x + direction.x) + (y + direction.y) * staggeredDim.x + (z + direction.z) * staggeredDim.x * staggeredDim.y];
		velFeild[(staggeredDim.x-1) + (staggeredDim.y - 1) * staggeredDim.x + (staggeredDim.z - 1) * staggeredDim.x * staggeredDim.y] = velFeild[((staggeredDim.x - 1) - direction.x) + ((staggeredDim.y - 1) - direction.y) * staggeredDim.x + ((staggeredDim.z - 1) - direction.z) * staggeredDim.x * staggeredDim.y];
	}
}


__device__ float avgU(float* vel, int x, int y, int z, uint3 staggeredDim) {
	float u = 
		vel[x + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x + 1) + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
	return u / 8;
}

__device__ float avgV(float* vel, int x, int y, int z, uint3 staggeredDim) {
	float v = 
		vel[x + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + (y + 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y + 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];

	return v / 8;
}

__device__ float avgW(float* vel, int x, int y, int z, uint3 staggeredDim) {
	float w =
		vel[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y - 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] +
		vel[x + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + y * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[x + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y] +
		vel[(x - 1) + (y - 1) * staggeredDim.x + (z-1) * staggeredDim.x * staggeredDim.y];
	return w / 8;
}



__device__ float sampleSmoke(float* field, uint3 gridDim, float3 pos, float3 delta, uint3 normalDim) {
	
	float x = fmaxf(fminf(pos.x, normalDim.x-1), 1);
	float y = fmaxf(fminf(pos.y, normalDim.y-1), 1);
	float z = fmaxf(fminf(pos.z, normalDim.z-1), 1);

	//interpelation
	int x0 = (int)fminf(floorf(x - delta.x), normalDim.x-1);
	float xw1 = (x - delta.x) - x0;
	float xw0 = 1 - xw1;
	int x1 = (int)fminf(x0 + 1, normalDim.x - 1);

	int y0 = (int)fminf(floorf(y - delta.y), normalDim.y-1);
	float yw1 = (y - delta.y) - y0;
	float yw0 = 1 - yw1;
	int y1 = (int)fminf(y0 + 1, normalDim.y - 1);

	int z0 = (int)fminf(floorf(z - delta.z), normalDim.z-1);
	float zw1 = (z - delta.z) - z0;
	float zw0 = 1 - zw1;
	int z1 = (int)fminf(z0 + 1, normalDim.z - 1);

	return
		xw0 * yw0 * zw0 * field[x0 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw0 * field[x1 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw0 * field[x0 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw0 * field[x1 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw0 * zw1 * field[x0 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw1 * field[x1 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw1 * field[x0 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw1 * field[x1 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y];
		

}

__device__ float sampleVelocity(float* field, uint3 gridDim, float3 pos, float3 delta, uint3 normalDim) {

	float x = fmaxf(fminf(pos.x, gridDim.x - 1), 1);
	float y = fmaxf(fminf(pos.y, gridDim.y - 1), 1);
	float z = fmaxf(fminf(pos.z, gridDim.z - 1), 1);

	//interpelation
	int x0 = (int)fminf(floorf(x - delta.x), gridDim.x - 1);
	float xw1 = (x - delta.x) - x0;
	float xw0 = 1 - xw1;
	int x1 = (int)fminf(x0 + 1, gridDim.x - 1);

	int y0 = (int)fminf(floorf(y - delta.y), gridDim.y - 1);
	float yw1 = (y - delta.y) - y0;
	float yw0 = 1 - yw1;
	int y1 = (int)fminf(y0 + 1, gridDim.y - 1);

	int z0 = (int)fminf(floorf(z - delta.z), gridDim.z - 1);
	float zw1 = (z - delta.z) - z0;
	float zw0 = 1 - zw1;
	int z1 = (int)fminf(z0 + 1, gridDim.z - 1);

	return
		xw0 * yw0 * zw0 * field[x0 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw0 * field[x1 + y0 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw0 * field[x0 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw0 * field[x1 + y1 * gridDim.x + z0 * gridDim.x * gridDim.y] +
		xw0 * yw0 * zw1 * field[x0 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw0 * zw1 * field[x1 + y0 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw0 * yw1 * zw1 * field[x0 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y] +
		xw1 * yw1 * zw1 * field[x1 + y1 * gridDim.x + z1 * gridDim.x * gridDim.y];


}







__global__ void velocityAdvectionU(float *u0, float *u1, float *v0, float *w0, bool* s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x && y < normalDim.y-1 && z < normalDim.z-1) {
		
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[(x - 1) + y * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			float x_t = x;
			float y_t = y + 0.5;
			float z_t = z + 0.5; 

			float u = u0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
			float v = avgV(v0, x, y, z, staggeredDim);
			float w = avgW(w0, x, y, z, staggeredDim);

			x_t = x_t - dt * u;
			y_t = y_t - dt * v;
			z_t = z_t - dt * w;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0, 0.5, 0.5 };

			float newU = sampleSmoke(u0, staggeredDim, pos, delta, normalDim);
			u1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = newU;

		}
	}
}

__global__ void velocityAdvectionV(float* v0, float* v1, float* u0, float* w0, bool* s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x-1 && y < normalDim.y && z < normalDim.z - 1) {

		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[x + (y-1) * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			float x_t = x + 0.5;
			float y_t = y;
			float z_t = z + 0.5;

			float u = avgU(u0, x, y, z, staggeredDim);
			float v = v0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];
			float w = avgW(w0, x, y, z, staggeredDim);

			x_t = x_t - dt * u;
			y_t = y_t - dt * v;
			z_t = z_t - dt * w;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0.5, 0, 0.5 };

			float newU = sampleSmoke(v0, staggeredDim, pos, delta, normalDim);
			v1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = newU;

		}
	}
}

__global__ void velocityAdvectionW(float* w0, float* w1, float* u0, float* v0, bool* s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if (x < normalDim.x - 1 && y < normalDim.y - 1 && z < normalDim.z) {

		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0 && s[x + y * normalDim.x + (z-1) * normalDim.x * normalDim.y] != 0) {
			float x_t = x + 0.5;
			float y_t = y + 0.5;
			float z_t = z;

			float u = avgU(u0, x, y, z, staggeredDim);
			float v = avgV(v0, x, y, z, staggeredDim);
			float w = w0[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y];

			x_t = x_t - dt * u;
			y_t = y_t - dt * v;
			z_t = z_t - dt * w;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0.5, 0.5, 0 };

			float newU = sampleSmoke(w0, staggeredDim, pos, delta, normalDim);
			w1[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] = newU;

		}
	}
}

__global__ void advectSmoke(float* smoke0, float *smoke1, float* u, float* v, float* w, bool *s, uint3 normalDim, uint3 staggeredDim, float dt) {
	int x = threadIdx.x + blockDim.x * blockIdx.x+1;
	int y = threadIdx.y + blockDim.y * blockIdx.y+1;
	int z = threadIdx.z + blockDim.z * blockIdx.z+1;

	if (x < normalDim.x-1 && y < normalDim.y-1 && z < normalDim.z-1) {
		if (s[x + y * normalDim.x + z * normalDim.x * normalDim.y] != 0) {
			float u_t = (u[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + u[(x + 1) + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y]) / 2;
			float v_t = (v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[x + (y + 1) * staggeredDim.x + z * staggeredDim.x * staggeredDim.y]) / 2;
			float w_t = (w[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + w[x + y * staggeredDim.x + (z + 1) * staggeredDim.x * staggeredDim.y]) / 2;

			float x_t = x + 0.5 - u_t * dt;
			float y_t = y + 0.5 - v_t * dt;
			float z_t = z + 0.5 - w_t * dt;

			float3 pos = { x_t, y_t, z_t };
			float3 delta = { 0.5, 0.5, 0.5 };

			smoke1[x + y * normalDim.x + z * normalDim.x * normalDim.y] = sampleSmoke(smoke0, normalDim, pos, delta, normalDim); 
		}
	}
}

__global__ void copyFeildAToB(float* feildA, float* feildB, uint3 dim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x + 1;
	int y = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int z = threadIdx.z + blockDim.z * blockIdx.z + 1;

	if(x < dim.x && y < dim.y && z < dim.z) {
		feildB[x + y * dim.x + z * dim.x * dim.y] = feildA[x + y * dim.x + z * dim.x * dim.y];
	}
}

__global__ void visualizeV(float* v, float* smoke, uint3 normalDim, uint3 staggeredDim, uint3 dir) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] =
			(v[x + y * staggeredDim.x + z * staggeredDim.x * staggeredDim.y] + v[(x+dir.x) + (y + dir.y) * staggeredDim.x + (z+dir.z) * staggeredDim.x * staggeredDim.y]) / 2;
	}
}

__global__ void visualizeS(bool *s, float* smoke, uint3 normalDim, uint3 staggeredDim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] = (float)s[x + y * normalDim.x + z * normalDim.x * normalDim.y];
	}
}


__global__ void visualizeP(float *p, float* smoke, uint3 normalDim, uint3 staggeredDim) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < normalDim.x && y < normalDim.y && z < normalDim.z) {
		smoke[x + y * normalDim.x + z * normalDim.x * normalDim.y] = p[x + y * normalDim.x + z * normalDim.x * normalDim.y];
	}
}


__global__ void smokeKernal(float *grid, int width, int height, int depth) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	int z = threadIdx.z + blockDim.z * blockIdx.z;

	if (x < width && y < height && z < depth) {
		if (y < height - 1) {
			if (grid[x + (y + 1) * width + z * width * height] > 0.0f) {
				grid[x + y * width + z * width * height] = 1.0f;
			}
			else {
				grid[x + y * width + z * width * height] = 0.0f;
			}
		}
		else {
			grid[x + y * width + z * width * height] = 0.0f;
		}
		
		//grid[4] = 1.0f;
		//grid[8] = 1.0f;
		//grid[27] = 1.0f;
	}
}

int tempIndexNow = 1;
int tempIndexPast = 0;
int velindex = 0;




void drawObjects() {


	//draw smokeSources
	std::vector<Sphere> smokeSources = getObjectsOfType(1);
	std::vector<float> smokeData;
	for (int i = 0; i < smokeSources.size(); i++) {
		smokeData.push_back(smokeSources[i].pos.x);
		smokeData.push_back(smokeSources[i].pos.y);
		smokeData.push_back(smokeSources[i].pos.z);
		smokeData.push_back(smokeSources[i].radius);
	}

	hipError_t err;

	err = hipMemcpy(dev_smokeSources, smokeData.data(), smokeData.size() * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	dim3 dimBlock(8, 8, 8); //512 
	dim3 dimGrid(ceil(smokeDim.x / 8.0), ceil(smokeDim.y / 8.0), ceil(smokeDim.z / 8.0));


	fillSmoke << <dimGrid, dimBlock >> > (dev_smoke[0], dev_smoke[1], smokeDim, dev_smokeSources, smokeSources.size());

	smokeSources.clear();
	smokeData.clear();




	//draw obstacles
	std::vector<Sphere> obstacles = getObjectsOfType(0); 
	std::vector<float> obstaclesData;
	for (int i = 0; i < obstacles.size(); i++) {
		obstaclesData.push_back(obstacles[i].pos.x);
		obstaclesData.push_back(obstacles[i].pos.y);
		obstaclesData.push_back(obstacles[i].pos.z);
		obstaclesData.push_back(obstacles[i].vel.x);
		obstaclesData.push_back(obstacles[i].vel.y);
		obstaclesData.push_back(obstacles[i].vel.z);
		obstaclesData.push_back(obstacles[i].radius);
	}

	err = hipMemcpy(dev_obstacles, obstaclesData.data(), obstaclesData.size() * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from host to device: %s\n", hipGetErrorString(err));
		exit(-1);
	}

	fillObstacle << <dimGrid, dimBlock >> > (dev_s, smokeDim, dev_obstacles, obstacles.size());

	obstacles.clear();
	obstaclesData.clear();

}


void simulate(float* smoke_grid, float dt) {
	hipError_t err;

	tempIndexPast = tempIndexNow;
	if (tempIndexNow == 0) tempIndexNow = 1;
	else tempIndexNow = 0;


	drawObjects();


	
	dim3 dimBlock(8, 8, 8); //512 
	dim3 dimGrid(ceil(smokeDim.x / 8.0), ceil(smokeDim.y / 8.0), ceil(smokeDim.z / 8.0));
	
	integrate<<<dimGrid, dimBlock>>>(dev_v[smokeIndex], dev_smoke[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt, gravity, buoyancy_alpha);

	velocityConfinement << <dimGrid, dimBlock >> > (dev_u[tempIndexNow], dev_v[tempIndexNow], dev_w[tempIndexNow], smokeDim, smokeStaggeredDim, dt);



	dim3 divdimBlock(8, 8, 8); //512 
	dim3 divdimGrid(ceil(ceil(smokeDim.x / 2.0) / 8.0), ceil(smokeDim.y / 8.0), ceil(smokeDim.z / 8.0));
	int num_iterations = 30;
	for (int i = 0; i < num_iterations; i++) {
		divergence << <divdimGrid, divdimBlock >> > (dev_u[tempIndexNow], dev_v[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, 0);
		divergence << <divdimGrid, divdimBlock >> > (dev_u[tempIndexNow], dev_v[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, 1);
	}



	velocityAdvectionU << <dimGrid, dimBlock >> > (dev_u[tempIndexNow], dev_u[tempIndexPast], dev_v[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt);
	velocityAdvectionV << <dimGrid, dimBlock >> > (dev_v[tempIndexNow], dev_v[tempIndexPast], dev_u[tempIndexNow], dev_w[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt);
	velocityAdvectionW << <dimGrid, dimBlock >> > (dev_w[tempIndexNow], dev_w[tempIndexPast], dev_u[tempIndexNow], dev_v[tempIndexNow], dev_s, smokeDim, smokeStaggeredDim, dt);


	advectSmoke << <dimGrid, dimBlock >> > (dev_smoke[tempIndexNow], dev_smoke[tempIndexPast], dev_u[tempIndexPast], dev_v[tempIndexPast], dev_w[tempIndexPast], dev_s, smokeDim, smokeStaggeredDim, dt);

	uint3 dir = { 0,1,0 };

	err = hipMemcpy(smoke_grid, dev_smoke[tempIndexPast], smokeDim.x * smokeDim.y * smokeDim.z * sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Error copying data from device to host: %s\n", hipGetErrorString(err));
		exit(-1);
	}
}

